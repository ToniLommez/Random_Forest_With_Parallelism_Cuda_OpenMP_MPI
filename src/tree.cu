#include "hip/hip_runtime.h"
#include "../include/tree.hpp"
#include <hip/hip_runtime.h>
#include <>
#include <limits>
#include <cuda/std/limits>
#include <iostream>

#ifdef CUDA

__device__ __forceinline__ float atomicMinFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMin((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMax((unsigned int *)addr, __float_as_uint(value)));

    return old;
}

__global__ void compute_gini_parallel(
    const float* d_X, const float* d_y,
    int num_samples, int num_features,
    float* d_best_impurity, int* d_best_feature, float* d_best_threshold) {

    // if(threadIdx.x == 0 && blockIdx.x == 0) {
    //     printf("device: num_samples: %d\ndevice: num_features: %d\n\n", num_samples, num_features);
    // }

    extern __shared__ float shared_memory[]; // For reduction
    float* shared_ginis = shared_memory;    // Gini values
    int* shared_features = (int*)&shared_ginis[blockDim.x]; // Features
    float* shared_thresholds = (float*)&shared_features[blockDim.x]; // Thresholds

    int feature = blockIdx.x; // Each block works on one feature
    int thread_idx = threadIdx.x;

    float best_local_impurity = cuda::std::numeric_limits<float>::max();
    float best_local_threshold = 0.0f;

    // Iterate over thresholds (samples) assigned to this thread
    for (int i = thread_idx; i < num_samples; i += blockDim.x) {
        float threshold = d_X[i * num_features + feature]; // Get threshold

        // Split data (device-side operations)
        int left_count = 0, right_count = 0;
        float left_gini = 0.0f, right_gini = 0.0f;

        for (int j = 0; j < num_samples; ++j) {
            float value = d_X[j * num_features + feature];
            if (value <= threshold) {
                left_count++;
                left_gini += d_y[j];
            } else {
                right_count++;
                right_gini += d_y[j];
            }
        }

        left_gini = (left_gini / left_count) * (1.0f - (left_gini / left_count));
        right_gini = (right_gini / right_count) * (1.0f - (right_gini / right_count));

        float weighted_impurity =
            (left_count * left_gini + right_count * right_gini) / num_samples;

        // Update local best impurity
        if (weighted_impurity < best_local_impurity) {
            best_local_impurity = weighted_impurity;
            best_local_threshold = threshold;
        }
    }

    // Store local results in shared memory
    shared_ginis[thread_idx] = best_local_impurity;
    shared_thresholds[thread_idx] = best_local_threshold;
    shared_features[thread_idx] = feature;

    __syncthreads();

    // Reduction: Find best impurity among threads in the block
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (thread_idx < stride) {
            if (shared_ginis[thread_idx + stride] < shared_ginis[thread_idx]) {
                shared_ginis[thread_idx] = shared_ginis[thread_idx + stride];
                shared_thresholds[thread_idx] = shared_thresholds[thread_idx + stride];
            }
        }
        __syncthreads();
    }

    // Block-level result: store the best result of this block
    if (thread_idx == 0) {
        if (shared_ginis[0] < *d_best_impurity) {
            // atomicMinFloat(d_best_impurity, shared_ginis[0]);
            *d_best_impurity = shared_ginis[0];
            *d_best_feature = shared_features[0];
            *d_best_threshold = shared_thresholds[0];
        }
    }
}

void cuda_best_threshold_sender(
    const float_matrix& X, const float_vector& y,
    int num_samples, int num_features, int* best_feature, float* best_threshold) {

    // hipDeviceProp_t prop;
    // hipGetDeviceProperties(&prop, 0);
    // std::cout << "Shared memory per block: " << prop.sharedMemPerBlock << " bytes" << std::endl;

    // Flatten X for device memory
    std::vector<float> X_flattened;
    for (const auto& row : X) {
        X_flattened.insert(X_flattened.end(), row.begin(), row.end());
    }

    // Allocate device memory
    float* d_X;
    float* d_y;
    float* d_best_impurity;
    int* d_best_feature;
    float* d_best_threshold;

    hipMalloc(&d_X, num_samples * num_features * sizeof(float));
    hipMalloc(&d_y, num_samples * sizeof(float));
    hipMalloc(&d_best_impurity, sizeof(float));
    hipMalloc(&d_best_feature, sizeof(int));
    hipMalloc(&d_best_threshold, sizeof(float));

    // Copy data to device
    hipMemcpy(d_X, X_flattened.data(), num_samples * num_features * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y.data(), num_samples * sizeof(float), hipMemcpyHostToDevice);

    // Initialize best impurity on the device
    float best_impurity = numeric_limits<float>::max();
    hipMemcpy(d_best_impurity, &best_impurity, sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threads_per_block = 1024;
    int shared_memory_size = threads_per_block * (sizeof(float) + sizeof(int) + sizeof(float));
    // std::cout << "Shared memory calculated: " << shared_memory_size << " bytes" << std::endl;
    // std::cout << "host: num_samples: " << num_samples << std::endl;
    // std::cout << "host: num_features: " << num_features << std::endl;
    compute_gini_parallel<<<num_features, threads_per_block, shared_memory_size>>>(
        d_X, d_y, num_samples, num_features, d_best_impurity, d_best_feature, d_best_threshold);

    // Copy results back to host
    hipMemcpy(best_feature, d_best_feature, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(best_threshold, d_best_threshold, sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_X);
    hipFree(d_y);
    hipFree(d_best_impurity);
    hipFree(d_best_feature);
    hipFree(d_best_threshold);
}

#endif
